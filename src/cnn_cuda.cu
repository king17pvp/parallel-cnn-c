#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "cnn.h"
#include "utils.h"
#include "cnn_cuda.h"

#define CUDA_CHECK(err) do { \
    if (err != hipSuccess) { \
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err)); \
        exit(EXIT_FAILURE); \
    } \
} while(0)

__device__ float leaky_relu_cuda(float x) {
    return x > 0 ? x : 0.01f * x;
}

__device__ float relu_cuda(float x) {
    return x > 0 ? x : 0;
}

__global__ void conv_forward_kernel(const float *input, const float *weights, const float *biases,
                                    float *output, int in_width, int in_height, int in_channel,
                                    int out_width, int out_height, int out_channel, int kernel_size) {
    int out_c = blockIdx.z;
    int out_x = blockIdx.x * blockDim.x + threadIdx.x;
    int out_y = blockIdx.y * blockDim.y + threadIdx.y;

    if (out_x < out_width && out_y < out_height && out_c < out_channel) {
        float sum = 0.0f;
        for (int in_c = 0; in_c < in_channel; ++in_c) {
            for (int ky = 0; ky < kernel_size; ++ky) {
                for (int kx = 0; kx < kernel_size; ++kx) {
                    int in_x = out_x + kx;
                    int in_y = out_y + ky;
                    int in_idx = in_c * in_height * in_width + in_y * in_width + in_x;
                    int w_idx = out_c * in_channel * kernel_size * kernel_size + in_c * kernel_size * kernel_size + ky * kernel_size + kx;
                    sum += input[in_idx] * weights[w_idx];
                }
            }
        }
        int out_idx = out_c * out_height * out_width + out_y * out_width + out_x;
        output[out_idx] = relu_cuda(sum + biases[out_c]);
    }
}

Tensor3D conv_forward_cuda(Tensor3D input, ConvLayer *layer) {
    int out_w = input.width - layer->kernel_size + 1;
    int out_h = input.height - layer->kernel_size + 1;
    int out_c = layer->out_channels;
    int in_c = layer->in_channels;
    int ks = layer->kernel_size;
    int input_size = input.width * input.height * input.channels;
    int output_size = out_w * out_h * out_c;
    int weight_size = out_c * in_c * ks * ks;

    float *d_input, *d_weights, *d_biases, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, sizeof(float) * input_size));
    CUDA_CHECK(hipMalloc(&d_weights, sizeof(float) * weight_size));
    CUDA_CHECK(hipMalloc(&d_biases, sizeof(float) * out_c));
    CUDA_CHECK(hipMalloc(&d_output, sizeof(float) * output_size));

    CUDA_CHECK(hipMemcpy(d_input, input.data, sizeof(float) * input_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_weights, layer->weights, sizeof(float) * weight_size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_biases, layer->biases, sizeof(float) * out_c, hipMemcpyHostToDevice));

    dim3 threads(16, 16);
    dim3 blocks((out_w + 15) / 16, (out_h + 15) / 16, out_c);
    conv_forward_kernel<<<blocks, threads>>>(d_input, d_weights, d_biases, d_output,
                                             input.width, input.height, in_c,
                                             out_w, out_h, out_c, ks);
    CUDA_CHECK(hipDeviceSynchronize());

    float *output_data = (float*)malloc(sizeof(float) * output_size);
    CUDA_CHECK(hipMemcpy(output_data, d_output, sizeof(float) * output_size, hipMemcpyDeviceToHost));

    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_biases);
    hipFree(d_output);

    return (Tensor3D){ out_w, out_h, out_c, output_data };
}

__global__ void maxpool_forward_kernel(const float *input, float *output,
                                       int in_w, int in_h, int in_c,
                                       int pool_size, int out_w, int out_h) {
    int c = blockIdx.z;
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    if (i < out_h && j < out_w && c < in_c) {
        float max_val = -1e9;
        for (int pi = 0; pi < pool_size; ++pi) {
            for (int pj = 0; pj < pool_size; ++pj) {
                int y = i * pool_size + pi;
                int x = j * pool_size + pj;
                int idx = c * in_h * in_w + y * in_w + x;
                max_val = fmaxf(max_val, input[idx]);
            }
        }
        int out_idx = c * out_h * out_w + i * out_w + j;
        output[out_idx] = max_val;
    }
}

Tensor3D maxpool_forward_cuda(Tensor3D input, int pool_size) {
    int in_w = input.width;
    int in_h = input.height;
    int in_c = input.channels;
    int out_w = in_w / pool_size;
    int out_h = in_h / pool_size;

    float *d_input, *d_output;
    size_t in_size = in_w * in_h * in_c * sizeof(float);
    size_t out_size = out_w * out_h * in_c * sizeof(float);

    CUDA_CHECK(hipMalloc(&d_input, in_size));
    CUDA_CHECK(hipMemcpy(d_input, input.data, in_size, hipMemcpyHostToDevice));

    CUDA_CHECK(hipMalloc(&d_output, out_size));

    dim3 blockSize(16, 16);
    dim3 gridSize((out_w + 15) / 16, (out_h + 15) / 16, in_c);
    maxpool_forward_kernel<<<gridSize, blockSize>>>(d_input, d_output, in_w, in_h, in_c, pool_size, out_w, out_h);

    CUDA_CHECK(hipDeviceSynchronize());

    float *h_output = (float *)malloc(out_size);
    CUDA_CHECK(hipMemcpy(h_output, d_output, out_size, hipMemcpyDeviceToHost));

    hipFree(d_input);
    hipFree(d_output);
    free(input.data);  // optional if no reuse

    return (Tensor3D){out_w, out_h, in_c, h_output};
}


Vector flatten_cuda(Tensor3D input) {
    int total = input.width * input.height * input.channels;
    float *flat = input.data; // Directly reuse
    return (Vector){ total, flat };
}

__global__ void fc_forward_kernel(const float *input, const float *weights, const float *biases,
                                  float *output, int in_features, int out_features) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < out_features) {
        float sum = 0.0f;
        for (int j = 0; j < in_features; j++) {
            sum += weights[idx * in_features + j] * input[j];
        }
        output[idx] = relu_cuda(sum + biases[idx]);
    }
}

Vector fc_forward_cuda(Vector input, FullyConnectedLayer *layer) {
    int in_features = input.size;
    int out_features = layer->out_features;

    float *d_input, *d_weights, *d_biases, *d_output;
    CUDA_CHECK(hipMalloc(&d_input, sizeof(float) * in_features));
    CUDA_CHECK(hipMalloc(&d_weights, sizeof(float) * in_features * out_features));
    CUDA_CHECK(hipMalloc(&d_biases, sizeof(float) * out_features));
    CUDA_CHECK(hipMalloc(&d_output, sizeof(float) * out_features));

    CUDA_CHECK(hipMemcpy(d_input, input.data, sizeof(float) * in_features, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_weights, layer->weights, sizeof(float) * in_features * out_features, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_biases, layer->biases, sizeof(float) * out_features, hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = (out_features + threads - 1) / threads;
    fc_forward_kernel<<<blocks, threads>>>(d_input, d_weights, d_biases, d_output,
                                           in_features, out_features);
    CUDA_CHECK(hipDeviceSynchronize());

    float *output_data = (float*)malloc(sizeof(float) * out_features);
    CUDA_CHECK(hipMemcpy(output_data, d_output, sizeof(float) * out_features, hipMemcpyDeviceToHost));

    hipFree(d_input);
    hipFree(d_weights);
    hipFree(d_biases);
    hipFree(d_output);

    free(input.data);
    return (Vector){ out_features, output_data };
}

void cnn_forward_cuda(CNN *cnn) {
    int total_input = cnn->input_width * cnn->input_height * cnn->input_channels;
    float *copy = (float*)malloc(sizeof(float) * total_input);
    for (int i = 0; i < total_input; i++) copy[i] = cnn->input_data[i];

    Tensor3D x = { cnn->input_width, cnn->input_height, cnn->input_channels, copy };

    for (int i = 0; i < cnn->num_conv_layers; i++) {
        x = conv_forward_cuda(x, &cnn->conv_layers[i]);
        // x = maxpool_forward_cuda(x, 2);
    }

    Vector v = flatten_cuda(x);
    for (int i = 0; i < cnn->num_fc_layers; i++) {
        // int apply_activation = (i != cnn->num_fc_layers - 1);
        v = fc_forward_cuda(v, &cnn->fc_layers[i]);
    }
    cnn->output = v.data[0];

    free(v.data);
}